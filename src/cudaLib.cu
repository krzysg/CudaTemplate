#include "hip/hip_runtime.h"
#include "cudaLib.h"

#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace {
    template <typename T>
    __global__ void kernel(T aValue) {
        printf("Hello from CUDA with value=%d\n", aValue);
    }
}

// explicit instantiation
template void cudaRun<int>(int);
template void cudaRun<uint16_t>(uint16_t);

template <typename T>
void cudaRun(T aValue) {
    kernel<<<1,1>>>(aValue);
    hipDeviceSynchronize();
    if (hipGetLastError() != hipSuccess) printf("Error: %s\n", hipGetErrorString(hipGetLastError()));
}
