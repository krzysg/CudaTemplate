#include "hip/hip_runtime.h"
#include <iostream>
#include "tools.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace {
void foo() {
	run<int>();
}

template <typename T>
__global__ void kernel(T val) {
	printf("Hello CUDA. %d\n", val);
}
}

template <typename T>
void run() {
	T i = 3;
       	kernel<<<1,1>>>(i);
        hipDeviceSynchronize();
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
}

